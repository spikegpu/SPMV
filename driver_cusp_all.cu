#include <iostream>
#include <cstdio>
#include <string>
#include <stdlib.h>
#include <cstring>
#include <cmath>
#include <src/timer.h>
#include <algorithm>

#include <cusp/io/matrix_market.h>
#include <cusp/ell_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas/blas.h>

using std::string;
using std::cout;
using std::cerr;
using std::endl;


typedef double REAL;

typedef typename cusp::coo_matrix<int, REAL, cusp::device_memory> CooMatrix;
typedef typename cusp::csr_matrix<int, REAL, cusp::device_memory> CsrMatrix;
typedef typename cusp::csr_matrix<int, REAL, cusp::host_memory>   CsrMatrixH;
typedef typename cusp::ell_matrix<int, REAL, cusp::device_memory> EllMatrix;
typedef typename cusp::hyb_matrix<int, REAL, cusp::device_memory> HybMatrix;
typedef typename cusp::array1d<REAL, cusp::device_memory>         Vector;
typedef typename cusp::array1d<REAL, cusp::host_memory>           VectorH;


int main (int argc, char **argv)
{
	int N, NNZ, NNZ2;

	if (argc < 2) return 1;

	string mat_file  = "/home/ali/CUDA_project/reordering/matrices/M_invD_";
	string mat_file2 = "/home/ali/CUDA_project/reordering/matrices/D_T_";
	string vec_file  = "/home/ali/CUDA_project/reordering/matrices/gamma_";

	mat_file.append(argv[1]);
	mat_file.append(".mtx");
	mat_file2.append(argv[1]);
	mat_file2.append(".mtx");
	vec_file.append(argv[1]);
	vec_file.append(".mtx");

	cout << argv[1] << endl;

	CsrMatrixH M_invD_h;
	CsrMatrixH D_T_h;

	cusp::io::read_matrix_market_file(M_invD_h, mat_file);
	cusp::io::read_matrix_market_file(D_T_h,    mat_file2);

	N    = M_invD_h.num_rows;
	NNZ  = M_invD_h.num_entries;
	NNZ2 = D_T_h.num_entries;

	Vector gamma;
	cusp::io::read_matrix_market_file(gamma, vec_file);

	Vector t1(N);
	Vector t2(N);

	{
		CsrMatrix M_invD, D_T;

		M_invD = M_invD_h;
		D_T    = D_T_h;

		CPUTimer loc_timer, timer2;
		double elapsed = 0.0;
		int counter = 10 - 1;
		loc_timer.Start();
		for (int i = 0; i < 10; i++) {
			if (i == 0)
				timer2.Start();

			cusp::multiply(M_invD, gamma, t1);
			cusp::multiply(D_T,    t1,    t2);
			hipDeviceSynchronize();

			if (i == 0)
				timer2.Stop();
		}
		loc_timer.Stop();
		elapsed = (loc_timer.getElapsed() - timer2.getElapsed())/ counter;
		cout << "CUSP CSR: " << elapsed << " " << 2.0 * (NNZ + NNZ2) / elapsed << endl;
	}

	{
		CooMatrix M_invD, D_T;

		M_invD = M_invD_h;
		D_T    = D_T_h;

		CPUTimer loc_timer, timer2;
		double elapsed = 0.0;
		int counter = 10 - 1;
		loc_timer.Start();
		for (int i = 0; i < 10; i++) {
			if (i == 0)
				timer2.Start();

			cusp::multiply(M_invD, gamma, t1);
			cusp::multiply(D_T,    t1,    t2);
			hipDeviceSynchronize();

			if (i == 0)
				timer2.Stop();
		}
		loc_timer.Stop();
		elapsed = (loc_timer.getElapsed() - timer2.getElapsed())/ counter;
		cout << "CUSP COO: " << elapsed << " " << 2.0 * (NNZ + NNZ2) / elapsed << endl;
	}

	{
		EllMatrix M_invD, D_T;

		M_invD = M_invD_h;
		D_T    = D_T_h;

		CPUTimer loc_timer, timer2;
		double elapsed = 0.0;
		int counter = 10 - 1;
		loc_timer.Start();
		for (int i = 0; i < 10; i++) {
			if (i == 0)
				timer2.Start();

			cusp::multiply(M_invD, gamma, t1);
			cusp::multiply(D_T,    t1,    t2);
			hipDeviceSynchronize();

			if (i == 0)
				timer2.Stop();
		}
		loc_timer.Stop();
		elapsed = (loc_timer.getElapsed() - timer2.getElapsed())/ counter;
		cout << "CUSP ELL: " << elapsed << " " << 2.0 * (NNZ + NNZ2) / elapsed << endl;
	}

	{
		HybMatrix M_invD, D_T;

		M_invD = M_invD_h;
		D_T    = D_T_h;

		CPUTimer loc_timer, timer2;
		double elapsed = 0.0;
		int counter = 10 - 1;
		loc_timer.Start();
		for (int i = 0; i < 10; i++) {
			if (i == 0)
				timer2.Start();

			cusp::multiply(M_invD, gamma, t1);
			cusp::multiply(D_T,    t1,    t2);
			hipDeviceSynchronize();

			if (i == 0)
				timer2.Stop();
		}
		loc_timer.Stop();
		elapsed = (loc_timer.getElapsed() - timer2.getElapsed())/ counter;
		cout << "CUSP HYB: " << elapsed << " " << 2.0 * (NNZ + NNZ2) / elapsed << endl;
	}
	return 0;
}
